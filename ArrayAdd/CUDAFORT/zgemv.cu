#include "hip/hip_runtime.h"
//-----------------------------------------------------------
// CUDA CPP double complex Array Add performance test
//-----------------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "cuda_settings.h"
#define IDX2F(i,j,ld) ((((j))*(ld))+((i)))
#define BLOCK_SIZE 32
#define BLOCK_SIZE_1D 1024

//void print_matrix(const int &m, const int &n, const hipDoubleComplex *A, const int &lda);
void createRandoms(int size, double *randomArray);

int main (int argc, char* argv[]){

    // CUDA_CHECK(hipGetDeviceProperties());

    int device = 0;
    CUDA_CHECK(hipSetDevice(device));

    hipStream_t stream;

    hipDoubleComplex *A;
    hipDoubleComplex *B, *C;
    hipDoubleComplex *d_A, *d_B, *d_C;
    int arraylen = 1000000;
    int nloop = 20;
    // int batchsize = 20000;
    if (argc > 1 ){
      arraylen = strtol(argv[1], nullptr, 0);
    }
    int N = arraylen;
    // double tol = exp10(-double(strtol(argv[1],nullptr,0)));
    A = (hipDoubleComplex *)malloc(arraylen*sizeof(hipDoubleComplex));
    B = (hipDoubleComplex *)malloc(arraylen*sizeof(hipDoubleComplex));
    C = (hipDoubleComplex *)malloc(arraylen*sizeof(hipDoubleComplex));
    //A = (double *)malloc(pow(N,2)*sizeof(double));
    double *rand1;
    double *rand2;
    rand1 = (double *)malloc(arraylen*sizeof(double));
    rand2 = (double *)malloc(arraylen*sizeof(double));
    
    // printf("Generating %d by %d random matrix... \n",N,N);
    createRandoms(N, rand1);
    createRandoms(N, rand2);
    for (int i=0;i<N;i++){
      B[i] = {rand1[i],rand2[i]};
    } 
    createRandoms(N, rand1);
    createRandoms(N, rand2);
    for (int i=0;i<N;i++){
      C[i] = {rand1[i],rand2[i]};
    } 

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamDefault));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), N*sizeof(hipDoubleComplex)));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), N*sizeof(hipDoubleComplex)));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C), N*sizeof(hipDoubleComplex)));

    CUDA_CHECK(hipMemset(d_A, 0, N*sizeof(hipDoubleComplex)));
    CUDA_CHECK(hipMemcpy(d_B, B, N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice ));
    CUDA_CHECK(hipMemcpy(d_C, C, N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice ));

    hipEvent_t start, stop;    
    float elapsed_time;
    double elapsed_time_sum;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    hipblasHandle_t blasHandle;
    static const hipDoubleComplex cone = {1.0, 0.0}, czero = {0.0, 0.0};

    CUBLAS_CHECK(hipblasCreate(&blasHandle));
    
    elapsed_time_sum=0.0;
    CUDA_CHECK(hipEventRecord(start, stream));
    for (int j=0; j<nloop; j++){ 
      // hipblasStatus_t hipblasZaxpy(hipblasHandle_t handle, int n,
      //                      const hipDoubleComplex *alpha,
      //                      const hipDoubleComplex *x, int incx,
      //                      hipDoubleComplex       *y, int incy)
      CUBLAS_CHECK(hipblasZaxpy(blasHandle, arraylen, &cone, d_B, 1, d_A, 1 ));      
      CUBLAS_CHECK(hipblasZaxpy(blasHandle, arraylen, &cone, d_C, 1, d_A, 1 ));            
    }
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipEventRecord(stop, stream));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    elapsed_time_sum+=elapsed_time;
    CUDA_CHECK(hipDeviceSynchronize());

    double ms2s=0.001;
    double avgfac=1.0/double(arraylen)/double(nloop);
    // printf("Avgfac test : %g %g %g %g \n", double(long(arraylen)*long(nloop)), 1/double(arraylen*nloop), 1.0/double(arraylen*nloop), 1.0/double(arraylen)/double(nloop));
    printf("Array Length: %d, nloop: %d, CUDA event time: %gs, avg time for each sum : %gs \n", arraylen, nloop, elapsed_time_sum/1000.0, elapsed_time_sum*ms2s*avgfac);
    // printf("Array Length: %d, nloop: %d, CUDA event time: %gs, avg time for each array 1 loop : %gs \n", arraylen, nloop, elapsed_time_sum/1000.0, elapsed_time_sum/1000.0/double(nloop));
    CUDA_CHECK(hipMemcpy(A, d_A, N*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost ));

    printf("The value of A[%d] is %30.15g + %30.15g i \n", 12000, A[11999].x, A[11999].y);
    /*
    */
}
