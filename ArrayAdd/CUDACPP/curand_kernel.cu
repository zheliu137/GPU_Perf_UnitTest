#include <cstdio>
#include <cstdlib>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hiprand.h>
//#include "cuda_settings.h"

void createRandoms(int size, double *h_randomArray){
    hiprandGenerator_t generator;
    double *randomArray;
    hipMalloc((void**)&randomArray, size*sizeof(double));
    hiprandCreateGenerator(&generator,HIPRAND_RNG_PSEUDO_XORWOW);
    hiprandSetPseudoRandomGeneratorSeed(generator,(int)time(NULL));
    hiprandGenerateUniformDouble(generator,randomArray,size);
    hipMemcpy(h_randomArray, randomArray, sizeof(double) * size , hipMemcpyDeviceToHost);
}

void createRandoms_gpu(int length_in_double, double *randomArray){
    hiprandGenerator_t generator;
    hiprandCreateGenerator(&generator,HIPRAND_RNG_PSEUDO_XORWOW);
    hiprandSetPseudoRandomGeneratorSeed(generator,(int)time(NULL));
    hiprandGenerateUniformDouble(generator,randomArray,length_in_double);
}
