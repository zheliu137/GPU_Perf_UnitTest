#include <math.h>
#include <cstdio>
#include <ctime>
#include <iostream>
#include <cstdlib>
#include <vector>
#include <complex>
#include <algorithm>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hipsolver.h>
    
#ifdef DEBUG
#define CUSOLVER_CHECK(err) (HandlecusolverError(err, __FILE__, __LINE__))
#define CUDA_CHECK(err) (HandleError(err, __FILE__, __LINE__))
#define CUBLAS_CHECK(err) (HandleBlasError(err, __FILE__, __LINE__))
#else
#define CUSOLVER_CHECK(err) (err)
#define CUDA_CHECK(err) (err)
#define CUBLAS_CHECK(err) (err)
#endif

static void HandleBlasError(hipblasStatus_t err, const char *file, int line)
{

    if (err != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "ERROR: %s in %s at line %d (error-code %d)\n",
                cublasGetStatusString(err), file, line, err);
        fflush(stdout);
        exit(-1);
    }
}



static void HandlecusolverError(hipsolverStatus_t err, const char *file, int line )
{

    if (err != HIPSOLVER_STATUS_SUCCESS)
    {
        fprintf(stderr, "ERROR: %d in %s at line %d, (error-code %d)\n",
                err, file, line, err);
        fflush(stdout);
        exit(-1);
    }
}

static void HandleError(hipError_t err, const char *file, int line)
{

    if (err != hipSuccess)
    {
        fprintf(stderr, "ERROR: %s in %s at line %d (error-code %d)\n",
                hipGetErrorString(err), file, line, err);
        fflush(stdout);
        exit(-1);
    }
}

template <typename T> void print_matrix(const int &m, const int &n, const T *A, const int &lda);

template <> void print_matrix(const int &m, const int &n, const float *A, const int &lda) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            std::printf("%0.2f ", A[j * lda + i]);
        }
        std::printf("\n");
    }
}

template <> void print_matrix(const int &m, const int &n, const double *A, const int &lda) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            std::printf("%0.2f ", A[j * lda + i]);
        }
        std::printf("\n");
    }
}

template <> void print_matrix(const int &m, const int &n, const hipComplex *A, const int &lda) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            std::printf("%0.2f + %0.2fj ", A[j * lda + i].x, A[j * lda + i].y);
        }
        std::printf("\n");
    }
}

template <>
void print_matrix(const int &m, const int &n, const hipDoubleComplex *A, const int &lda) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            std::printf("%0.2f + %0.2fj ", A[j * lda + i].x, A[j * lda + i].y);
        }
        std::printf("\n");
    }
}

    hipblasOperation_t char_to_cublas_trans(char trans)
    {
        hipblasOperation_t cuTrans;
        switch (trans)
        {
        case 'n':
        case 'N':
            cuTrans = HIPBLAS_OP_N;
            break;
        case 't':
        case 'T':
            cuTrans = HIPBLAS_OP_T;
            break;
        case 'c':
        case 'C':
            cuTrans = HIPBLAS_OP_C;
            break;
        default:
            exit(-1);
        }
        return cuTrans;
    }

    void matmul_strided_batched(int m, int n, int k, hipDoubleComplex *A, 
        hipDoubleComplex *B, hipDoubleComplex *C, int batch_count)
    {
        hipDoubleComplex alpha={1.0,0.0};
        hipDoubleComplex beta={0.0,0.0};

        long long int stridea=m*n;
        long long int strideb=n*k;
        long long int stridec=m*k;
        int lda=m;
        int ldb=n;
        int ldc=k;
        char transa = 'n', transb = 'n';
        hipDoubleComplex *dA = nullptr, *dB = nullptr, *dC = nullptr;
        hipblasHandle_t blasHandle;
        hipStream_t stream = NULL;

        CUDA_CHECK(hipMallocAsync((void **)&dA, batch_count * m * k * sizeof(hipDoubleComplex), stream));
        CUDA_CHECK(hipMallocAsync((void **)&dB, batch_count * k * n * sizeof(hipDoubleComplex), stream));
        CUDA_CHECK(hipMallocAsync((void **)&dC, batch_count * m * n * sizeof(hipDoubleComplex), stream));

        CUBLAS_CHECK(hipblasCreate(&blasHandle));
        CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
        CUBLAS_CHECK(hipblasSetStream(blasHandle, stream));

        CUDA_CHECK(hipMemcpyAsync(dA, A, batch_count * m * k * sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream));
        CUDA_CHECK(hipMemcpyAsync(dB, B, batch_count * k * n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream));
        int batch_size=batch_count;
        int nstep = 1;
        int batch_step=batch_size/nstep;
        // CUDA timer
        hipEvent_t start, stop;
        float elapsed_time;
        CUDA_CHECK(hipEventCreate(&start));
        CUDA_CHECK(hipEventCreate(&stop));

        for (int i=1;i<=nstep;i++){
            CUDA_CHECK(hipEventRecord(start,stream));
            
            batch_count=i*batch_step;
            int nloop = 1000000;
            for (int j=1; j<=nloop;j++){
            CUBLAS_CHECK(hipblasZgemmStridedBatched(blasHandle, char_to_cublas_trans(transa), char_to_cublas_trans(transb), m, n, k, &alpha, dA, lda, stridea, dB, ldb, strideb, &beta, dC, ldc, stridec, batch_count));
            }
            CUDA_CHECK(hipEventRecord(stop,stream));
            CUDA_CHECK(hipEventSynchronize(stop));
            CUDA_CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
            printf("batchsize: %d, CUDA event time: %gs, avg time per matmul : %g \n",batch_count*nloop,elapsed_time/1000.0,elapsed_time/1000.0/double(batch_count*nloop));
        }

        //CUDA_CHECK(cudaMemcpyAsync(C, dC, batch_count * m * n * sizeof(cuDoubleComplex), cudaMemcpyDeviceToHost, stream));

        CUDA_CHECK(hipStreamSynchronize(stream));

        CUDA_CHECK(hipFreeAsync(dA, stream));
        CUDA_CHECK(hipFreeAsync(dB, stream));
        CUDA_CHECK(hipFreeAsync(dC, stream));

        hipblasDestroy(blasHandle);
        CUDA_CHECK(hipStreamDestroy(stream));
    }
