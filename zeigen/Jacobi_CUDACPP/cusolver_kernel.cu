#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <complex>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include "cuda_settings.h"

//extern "C"
//{
//#define DEBUG
//#define SINGLERUN

int cusolver_c_batch(const int m, hipDoubleComplex *A_, const int nmat, const int batchSize, const double tol_) {

    const int lda = m;
    //const int nmat = 512;
    int nbatch = nmat/batchSize;

    hipsolverHandle_t cusolverH;
    hipStream_t stream;
    hipsolverSyevjInfo_t syevj_params;

    //std::vector<hipDoubleComplex> V(lda * m * nmat); // eigenvectors
    //std::vector<double> W(m*nmat);       // eigenvalues
    //printf("Allocate pinned memory.\n");
    printf("solving %d %dx%d matrices by Jacobi method, with tol = %g\n",nmat,m,m,tol_);
    hipDoubleComplex *A; // matrix stored in pinned memory
    CUDA_CHECK(hipHostMalloc((void **)&A,sizeof(hipDoubleComplex)*lda * m * batchSize));
    //hipDoubleComplex A[lda * m * nmat]; // matrix stored in pinned memory
    //hipDoubleComplex V[lda * m * nmat]; // eigenvectors
    //hipDoubleComplex AMV[m*nmat]; // A*V
    hipDoubleComplex *V; // eigenvectors
    hipDoubleComplex *AMV; // A*V
    V = (hipDoubleComplex *)malloc (lda * m * batchSize * sizeof (*V));
    AMV = (hipDoubleComplex *)malloc (m * batchSize * sizeof (*AMV));
    double W[m*batchSize];       // eigenvalues
    
    //printf("Copy matrix to pinned memory.\n");
    std::copy(A_,A_+lda * m * batchSize,A);
    hipDoubleComplex *d_A;
    double *d_W;
    int *devInfo;
    hipDoubleComplex *d_work;
    int lwork;
    int info_gpu[batchSize];

    /* configuration of syevj  */
    //const double tol = 1.e-12;
    double tol = tol_;
    const int max_sweeps = 40;
    const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvectors.
    const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    hipEvent_t start, stop;
    float elapsed_time;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    /* numerical results of syevj  */
    //double residual = 0;
    //int executed_sweeps = 0;
    int nloop = 1;
    for (int i=0;i<nloop;i++){ 
    // step 0: allocate device memory
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(hipDoubleComplex) * lda * m*batchSize));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_W), sizeof(double) * m * batchSize));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&devInfo), sizeof(int) * batchSize));

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

    /* step 2: configuration of syevj */
    CUSOLVER_CHECK(hipsolverDnCreateSyevjInfo(&syevj_params));

    /* default value of tolerance is machine zero */
    CUSOLVER_CHECK(hipsolverDnXsyevjSetTolerance(syevj_params, tol));

    /* default value of max. sweeps is 100 */
    CUSOLVER_CHECK(hipsolverDnXsyevjSetMaxSweeps(syevj_params, max_sweeps));

    /* step 3: copy A to device */
    CUDA_CHECK(
        hipMemcpyAsync(d_A, A, sizeof(hipDoubleComplex) * lda * m *batchSize, hipMemcpyHostToDevice, stream));

    /* step 4: query working space of syevj */
    CUSOLVER_CHECK(
          hipsolverDnZheevjBatched_bufferSize(cusolverH, jobz, uplo, m, 
          d_A, lda, d_W, &lwork, syevj_params,batchSize));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(hipDoubleComplex) * lwork)
);
    /* step 5: compute eigen-pair   */

    CUDA_CHECK(hipEventRecord(start,stream));


    for (int j=0; j<nbatch; j++){ 
    CUSOLVER_CHECK(hipsolverDnZheevjBatched(cusolverH, jobz, uplo, m, 
                    d_A, lda, d_W, d_work, lwork, devInfo, syevj_params, batchSize));
    }

    CUDA_CHECK(hipEventRecord(stop,stream));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&elapsed_time, start, stop));

    printf("batchsize: %d, nbatch: %d, CUDA event time: %gs, avg time per diag : %g \n",batchSize, nbatch,elapsed_time/1000.0,elapsed_time/1000.0/double(batchSize*nbatch));

    // step 6: check status, show eigenvalues, and eigenvectors 
      
    CUDA_CHECK(hipMemcpyAsync(V, d_A, sizeof(hipDoubleComplex) *lda*m *batchSize, 
                                                hipMemcpyDeviceToHost, stream));
    
    CUDA_CHECK(hipMemcpyAsync(W, d_W, sizeof(double)* m *batchSize, 
                                                hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipMemcpyAsync(&info_gpu, devInfo, sizeof(int) *batchSize, 
                                                hipMemcpyDeviceToHost, stream));
    for (int i = 0; i < batchSize; i++) {
        if (0 == info_gpu[i]) {
#ifdef SINGLERUN
            printf("matrix %d: syevj converges \n", i);
#endif
        } else if (0 > info_gpu[i]) {
            printf("Error: %d-th parameter is wrong \n", -info_gpu[i]);
            exit(1);
        } else { 
            printf("WARNING: matrix %d, info = %d : sygvj does not converge \n",
 i, info_gpu[i]);        
      }
#ifdef DEBUG
      printf("Eigenvalue =  (ascending order)\n");
      for (int j = i*lda; j < (i+1)*m; j++) {
        printf("W[%d] = %E\n", j - i*lda, W[j]);
      }

      printf("V = \n");
      print_matrix(m, m, &V[ lda * m * i ], lda);
      printf("=====\n");
#endif
}

    // step 7 free device memory

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_W));
    CUDA_CHECK(hipFree(devInfo));

    CUDA_CHECK(hipStreamDestroy(stream));
    CUSOLVER_CHECK(hipsolverDnDestroySyevjInfo(syevj_params));
    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

    }
    CUDA_CHECK(hipHostFree(A));
    CUDA_CHECK(hipDeviceReset());
    // step 8: check results
    double residual;
    for (int i=0; i < batchSize; i++ ) {
       residual = 0.0;
       for (int j=0;j < m; j++) { 
#ifdef DEBUG
           printf("A * V(%d), W(%d) * V (%d)\n",j,j,j);
#endif
           for (int k=0; k < m; k++) { 
               AMV[k] = {0.0,0.0};
               for (int l=0; l < m; l++) { 
                   AMV[k] = hipCadd(AMV[k],
                            hipCmul(A_[k+l*m], V[i*m*lda+l+j*m]));
               }
#ifdef DEBUG
               printf("%0.2f + %0.2fj ", AMV[k].x, AMV[k].y);
               printf("%0.2f + %0.2fj ", 
                     W[i*m+j]*V[i*m*lda+k+j*m].x, W[i*m+j]*V[i*m*lda+k+j*m].y);
               printf("\n");
#endif
               residual = residual + abs(AMV[k].x-W[i*m+j]*V[i*m*lda+k+j*m].x)+
                                     abs(AMV[k].y-W[i*m+j]*V[i*m*lda+k+j*m].y);
           }
       }
    }
#ifdef SINGLERUN
    printf("residual = %e \n", residual);
#endif
    
    return EXIT_SUCCESS;
}
