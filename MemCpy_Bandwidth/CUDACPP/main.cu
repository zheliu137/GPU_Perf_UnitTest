#include "hip/hip_runtime.h"
//Example 1. Application Using C and cuBLAS: 1-based indexing
//-----------------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "hipblas.h"
//#include "cuda_settings.h"

// void run_eig_wrapper_(const int N, hipDoubleComplex *x);
//void print_matrix(const int &m, const int &n, const hipDoubleComplex *A, const int &lda);
//void cusolver(int N); 
//void test(); 

int main (int argc, char* argv[]){
    hipDoubleComplex *A;
    //double *A;
    int nmat = 200000;
    int batchsize = 20000;
    if (argc > 1 ){
      N = strtol(argv[1],nullptr,0);
    }
    A = (hipDoubleComplex *)malloc(nmat*sizeof(hipDoubleComplex));


    // int device_count;

    // CUDA_CHECK(hipGetDeviceCount(&device_count));

    // // CUDA_CHECK(hipGetDeviceProperties());


    // current_device = 0;
    // while (current_device < device_count) {

    // }


    // findCudaDevice(argc, (const char **)argv)
    int device = 0;
    CUDA_CHECK(hipSetDevice(device));
v
    hipStream_t stream;

    // hipDoubleComplex *A;
    hipDoubleComplex *B, *C;
    hipDoubleComplex *d_A, *d_B, *d_C;
    int arraylen = 1000000;
    int nloop = 200;
    // int batchsize = 20000;
    if (argc > 1 ){
      arraylen = strtol(argv[1], nullptr, 0);
    }
    int N = arraylen;
    // double tol = exp10(-double(strtol(argv[1],nullptr,0)));
    // A = (hipDoubleComplex *)malloc(arraylen*sizeof(hipDoubleComplex));
    B = (hipDoubleComplex *)malloc(arraylen*sizeof(hipDoubleComplex));
    C = (hipDoubleComplex *)malloc(arraylen*sizeof(hipDoubleComplex));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamDefault));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), N*sizeof(hipDoubleComplex)));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), N*sizeof(hipDoubleComplex)));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C), N*sizeof(hipDoubleComplex)));


    hipEvent_t start, stop;    
    float elapsed_time;
    double elapsed_time_sum;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipMemset(d_A, 0, N*sizeof(hipDoubleComplex)));
    CUDA_CHECK(hipMemcpy(d_B, B, N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice ));
    CUDA_CHECK(hipMemcpy(d_C, C, N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice ));

    elapsed_time_sum=0.0;
    CUDA_CHECK(hipEventRecord(start, stream));
    // for (int j=0; j<nloop; j++){ 
    dim3 block_dim = BLOCK_SIZE_1D;
    // dim3 block_dim = 32;
    dim3 grid_dim;
    grid_dim.x = (arraylen+block_dim.x-1)/block_dim.x;
    printf("%d %d\n", grid_dim.x,block_dim.x);
    ArrayAdd<<<grid_dim, block_dim>>>((ComplexD*)d_A, (ComplexD*)d_B, (ComplexD*)d_C, arraylen, nloop);
    // ArrayAdd<<<grid_dim, block_dim>>>(arraylen, nloop);
    CUDA_CHECK(hipGetLastError());
    // }
    CUDA_CHECK(hipEventRecord(stop, stream));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    elapsed_time_sum+=elapsed_time;
    CUDA_CHECK(hipDeviceSynchronize());

    double ms2s=0.001;
    double avgfac=1.0/double(arraylen)/double(nloop);
    // printf("Avgfac test : %g %g %g %g \n", double(long(arraylen)*long(nloop)), 1/double(arraylen*nloop), 1.0/double(arraylen*nloop), 1.0/double(arraylen)/double(nloop));
    printf("Array Length: %d, nloop: %d, CUDA event time: %gs, avg time for each sum : %gs \n", arraylen, nloop, elapsed_time_sum/1000.0, elapsed_time_sum*ms2s*avgfac);




    // printf("Success.\n");

    }
