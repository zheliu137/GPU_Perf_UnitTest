//Example 1. Application Using C and cuBLAS: 1-based indexing
//-----------------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "hipblas.h"
//#include "cuda_settings.h"
#define IDX2F(i,j,ld) ((((j))*(ld))+((i)))

// void run_eig_wrapper_(const int N, cuDoubleComplex *x);
//void print_matrix(const int &m, const int &n, const cuDoubleComplex *A, const int &lda);
int  matmul_strided_batched(int n,int m,int k,
  hipDoubleComplex *A,hipDoubleComplex *B,hipDoubleComplex *C,int nmat);
void createRandoms(int size, double *randomArray);
//void cusolver(int N); 
//void test(); 

int main (int argc, char* argv[]){
    hipDoubleComplex *A;
    hipDoubleComplex *B;    
    hipDoubleComplex *C;    
    int N=32;
    if (argc > 1 ){
      N = strtol(argv[1],nullptr,0);
    }
    int nmat = 20000;
    A = (hipDoubleComplex *)malloc(pow(N,2)*sizeof(hipDoubleComplex)*nmat);
    B = (hipDoubleComplex *)malloc(pow(N,2)*sizeof(hipDoubleComplex)*nmat);
    C = (hipDoubleComplex *)malloc(pow(N,2)*sizeof(hipDoubleComplex)*nmat);
    int size=N; 
    double *rand1;
    double *rand2;
    
    rand1 = (double *)malloc(pow(size,2)*sizeof(double));
    rand2 = (double *)malloc(pow(size,2)*sizeof(double));
    printf("Generating %d by %d random matrix... \n",N,N);
    for (int l=0;l<nmat;l++){
    createRandoms(N, rand1);
    createRandoms(N, rand2);
    for (int i=0;i<N;i++){
    for (int j=0;j<N;j++){
      A[IDX2F(i,j,N)+l*N*N] = {rand1[i+j*N]+rand1[j+i*N],rand2[i+j*N]-rand2[j+i*N]};
      B[IDX2F(i,j,N)+l*N*N] = {rand1[i+j*N]+rand1[j+i*N],rand2[i+j*N]-rand2[j+i*N]};
    }
    } 
    } 
    
    // for (int i=0;i<N;i++){
    // for (int j=0;j<N;j++){
    //   //A[IDX2F(i,j,N)] = {double(i*j+1), double(3*i*j*(i-j)-j+i)};
    //   A[IDX2F(i,j,N)] = {double(i+j+1.0), 10.0*(i-j)};
    //   B[IDX2F(i,j,N)] = {double(i+j+1.0), 10.0*(i-j)};
    //   //A[IDX2F(i,j,N)] = float(i*j);
    //   //A[IDX2F(i,j,N)] = {1.0,1.2};
    //   //printf("%f\n",A[IDX2F(i,j,N)]);
    // }
    // }

    // printf("Success.\n");

    //cusolver_c_stream( N, A, nmat);
    matmul_strided_batched( N, N, N, A, B, C, nmat);

    }
